#include "hip/hip_runtime.h"
#include "goofit/Application.h"
#include "goofit/FitManager.h"
#include "goofit/UnbinnedDataSet.h"
#include "goofit/PDFs/LandauPdf.h"
#include "goofit/PDFs/NovosibirskPdf.h"
#include "goofit/PDFs/BifurGaussPdf.h"

#include "goofit/Variable.h"
#include "TH1F.h"
#include "TStyle.h"
#include "TCanvas.h"
#include "TRandom.h"

#include <sys/time.h>
#include <sys/times.h>
#include <iostream>

using namespace std;

// CPU-side Novosibirsk evaluation for use in generating toy MC.
double novosib(double x, double peak, double width, double tail) {
    double qa=0, qb=0, qc=0, qx=0, qy=0;

    if(fabs(tail) < 1.e-7)
        qc = 0.5*pow(((x-peak)/width), 2);
    else {
        qa = tail*sqrt(log(4.));
        qb = sinh(qa)/qa;
        qx = (x-peak)/width*qb;
        qy = 1.+tail*qx;

        //---- Cutting curve from right side

        if(qy > 1.E-7)
            qc = 0.5*(pow((log(qy)/tail), 2) + tail*tail);
        else
            qc = 15.0;
    }

    //---- Normalize the result

    return exp(-qc);
}

TCanvas* foo = 0;

void fitAndPlot(GooPdf* total, UnbinnedDataSet* data, TH1F& dataHist, Variable* xvar, const char* fname) {
    total->setData(data);
    FitManager fitter(total);
    fitter.fit();
    
    if(!fitter)
        std::exit(fitter);

    TH1F pdfHist("pdfHist", "", xvar->GetNumBins(), xvar->GetLowerLimit(), xvar->GetUpperLimit());
    pdfHist.SetStats(false);

    UnbinnedDataSet grid(xvar);
    double step = (xvar->GetUpperLimit() - xvar->GetLowerLimit())/xvar->GetNumBins();

    for(int i = 0; i < xvar->GetNumBins(); ++i) {
        xvar->value = xvar->GetLowerLimit() + (i + 0.5) * step;
        grid.addEvent();
    }

    total->setData(&grid);
    std::vector<std::vector<double>> pdfVals =  total->getCompProbsAtDataPoints();

    double totalPdf = 0;

    for(int i = 0; i < grid.getNumEvents(); ++i) {
        grid.loadEvent(i);
        pdfHist.Fill(xvar->value, pdfVals[0][i]);
        totalPdf += pdfVals[0][i];
    }

    for(int i = 0; i < xvar->GetNumBins(); ++i) {
        double val = pdfHist.GetBinContent(i+1);
        val /= totalPdf;
        val *= data->getNumEvents();
        pdfHist.SetBinContent(i+1, val);
    }

    //foo->SetLogy(true);
    dataHist.SetMarkerStyle(8);
    dataHist.SetMarkerSize(0.5);
    dataHist.Draw("p");
    pdfHist.SetLineColor(kBlue);
    pdfHist.SetLineWidth(3);
    pdfHist.Draw("lsame");
    foo->SaveAs(fname);
}

int main(int argc, char** argv) {
    GooFit::Application app("Simple fit example", argc, argv);

    try {
        app.run();
    } catch (const GooFit::ParseError &e) {
        return app.exit(e);
    }

    gStyle->SetCanvasBorderMode(0);
    gStyle->SetCanvasColor(10);
    gStyle->SetFrameFillColor(10);
    gStyle->SetFrameBorderMode(0);
    gStyle->SetPadColor(0);
    gStyle->SetTitleColor(1);
    gStyle->SetStatColor(0);
    gStyle->SetFillColor(0);
    gStyle->SetFuncWidth(1);
    gStyle->SetLineWidth(1);
    gStyle->SetLineColor(1);
    gStyle->SetPalette(1, 0);

    // Independent variable.
    Variable* xvar = new Variable("xvar", -100, 100);
    xvar->SetNumBins(1000); // For such a large range, want more bins for better accuracy in normalisation.

    // Data sets for the three fits.
    UnbinnedDataSet landdata(xvar);
    UnbinnedDataSet bifgdata(xvar);
    UnbinnedDataSet novodata(xvar);

    // Histograms for showing the fit.
    TH1F landHist("landHist", "", xvar->GetNumBins(), xvar->GetLowerLimit(), xvar->GetUpperLimit());
    TH1F bifgHist("bifgHist", "", xvar->GetNumBins(), xvar->GetLowerLimit(), xvar->GetUpperLimit());
    TH1F novoHist("novoHist", "", xvar->GetNumBins(), xvar->GetLowerLimit(), xvar->GetUpperLimit());
    landHist.SetStats(false);
    bifgHist.SetStats(false);
    novoHist.SetStats(false);

    TRandom donram(42);

    double maxNovo = 0;

    for(double x = xvar->GetLowerLimit(); x < xvar->GetUpperLimit(); x += 0.01) {
        double curr = novosib(x, 0.3, 0.5, 1.0);

        if(curr < maxNovo)
            continue;

        maxNovo = curr;
    }

    double leftSigma = 13;
    double rightSigma = 29;
    double leftIntegral = 0.5 / (leftSigma * sqrt(2*M_PI));
    double rightIntegral = 0.5 / (rightSigma * sqrt(2*M_PI));
    double totalIntegral = leftIntegral + rightIntegral;
    double bifpoint = -10;

    // Generating three sets of toy MC.
    for(int i = 0; i < 100000; ++i) {
        // Landau
        xvar->value = xvar->GetUpperLimit() + 1;

        while((xvar->value > xvar->GetUpperLimit()) || (xvar->value < xvar->GetLowerLimit())) {
            xvar->value = donram.Landau(20, 1);
        }

        landdata.addEvent();
        landHist.Fill(xvar->value);

        // Bifurcated Gaussian
        if(donram.Uniform() < (leftIntegral / totalIntegral)) {
            xvar->value = bifpoint - 1;

            while((xvar->value < bifpoint) || (xvar->value > xvar->GetUpperLimit()))
                xvar->value = donram.Gaus(bifpoint, rightSigma);
        } else {
            xvar->value = bifpoint + 1;

            while((xvar->value > bifpoint) || (xvar->value < xvar->GetLowerLimit()))
                xvar->value = donram.Gaus(bifpoint, leftSigma);
        }

        bifgdata.addEvent();
        bifgHist.Fill(xvar->value);

        // And Novosibirsk.
        while(true) {
            xvar->value = donram.Uniform(xvar->GetLowerLimit(), xvar->GetUpperLimit());
            double y = donram.Uniform(0, maxNovo);

            if(y < novosib(xvar->value, 0.3, 0.5, 1.0))
                break;
        }

        novodata.addEvent();
        novoHist.Fill(xvar->value);
    }

    foo = new TCanvas();

    Variable* mpv            = new Variable("mpv", 40, 0, 150);
    Variable* sigma          = new Variable("sigma", 5, 0, 30);
    GooPdf* landau = new LandauPdf("landau", xvar, mpv, sigma);
    fitAndPlot(landau, &landdata, landHist, xvar, "landau.png");


    Variable* nmean = new Variable("nmean", 0.4, -10.0, 10.0);
    Variable* nsigm = new Variable("nsigm", 0.6, 0.0, 1.0);
    Variable* ntail = new Variable("ntail", 1.1, 0.1, 0.0, 3.0);
    GooPdf* novo = new NovosibirskPdf("novo", xvar, nmean, nsigm, ntail);
    fitAndPlot(novo, &novodata, novoHist, xvar, "novo.png");

    Variable* gmean = new Variable("gmean", 3.0, 1, -15, 15);
    Variable* lsigm = new Variable("lsigm", 10, 1, 10, 20);
    Variable* rsigm = new Variable("rsigm", 20, 1, 10, 40);
    GooPdf* bifur = new BifurGaussPdf("bifur", xvar, gmean, lsigm, rsigm);
    fitAndPlot(bifur, &bifgdata, bifgHist, xvar, "bifur.png");

    return 0;
}
