#include "hip/hip_runtime.h"
#include "goofit/PDFs/basic/BWPdf.h"

namespace GooFit {

__device__ fptype device_BW(fptype *evt, ParameterContainer &pc) {
    fptype x      = evt[0];
    fptype mean   = pc.parameters[pc.parameterIdx + 1];
    fptype gamma  = pc.parameters[pc.parameterIdx + 2];
    fptype rootPi = -2. * atan2(-1.0, 0.0);
    fptype ret    = (gamma / ((x - mean) * (x - mean) + gamma * gamma / 4)) / (2 * rootPi);
    pc.incrementIndex (1, 2, 0, 0, 1);
    return ret;
}

__device__ device_function_ptr ptr_to_BW = device_BW;

__host__ BWPdf::BWPdf(std::string n, Variable *_x, Variable *mean, Variable *width)
    : GooPdf(_x, n) {
    std::vector<unsigned int> pindices;
    pindices.push_back(registerParameter(mean));
    pindices.push_back(registerParameter(width));
    GET_FUNCTION_ADDR(ptr_to_BW);
    initialize(pindices);
}

__host__ void BWPdf::recursiveSetIndices () {
    GOOFIT_TRACE("host_function_table[{}] = {}({})", num_device_functions, getName (), "ptr_to_BW");
    GET_FUNCTION_ADDR(ptr_to_BW);

    host_function_table[num_device_functions] = host_fcn_ptr;
    functionIdx = num_device_functions ++;

    populateArrays ();
}

} // namespace GooFit
