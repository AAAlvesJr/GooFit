#include "hip/hip_runtime.h"
#include "goofit/PDFs/basic/NovosibirskPdf.h"

namespace GooFit {

__device__ fptype device_Novosibirsk(fptype* evt, ParameterContainer &pc) {
    fptype _Mean  = pc.parameters[1];
    fptype _Sigma = pc.parameters[2];
    fptype _Tail  = pc.parameters[3];
    fptype x      = evt[0];

    pc.incrementIndex (1, 3, 0, 0, 1);

    fptype qa = 0;
    fptype qb = 0;
    fptype qc = 0;
    fptype qx = 0;
    fptype qy = 0;

    if(fabs(_Tail) < 1.e-7) {
        qc = 0.5 * POW2((x - _Mean) / _Sigma);
    } else {
        qa = _Tail * sqrt(log(4.));
        qb = sinh(qa) / qa;
        qx = (x - _Mean) / _Sigma * qb;
        qy = 1. + _Tail * qx;

        //---- Cutting curve from right side

        if(qy > 1.e-7)
            qc = 0.5 * (POW2(log(qy) / _Tail) + _Tail * _Tail);
        else
            qc = 15.0;
    }

    //---- Normalize the result
    return exp(-qc);
}

__device__ device_function_ptr ptr_to_Novosibirsk = device_Novosibirsk;

__host__ NovosibirskPdf::NovosibirskPdf(std::string n, Variable *_x, Variable *mean, Variable *sigma, Variable *tail)
    : GooPdf(_x, n) {
    std::vector<unsigned int> pindices;
    pindices.push_back(registerParameter(mean));
    pindices.push_back(registerParameter(sigma));
    pindices.push_back(registerParameter(tail));
    GET_FUNCTION_ADDR(ptr_to_Novosibirsk);
    initialize(pindices);
}

__host__ void NovosibirskPdf::recursiveSetIndices () {
    GOOFIT_TRACE("host_function_table[{}] = {}({})", num_device_functions, getName (), "ptr_to_Novosibirsk");
    GET_FUNCTION_ADDR(ptr_to_Novosibirsk);

    host_function_table[num_device_functions] = host_fcn_ptr;
    functionIdx = num_device_functions ++;

    populateArrays ();
}

} // namespace GooFit
